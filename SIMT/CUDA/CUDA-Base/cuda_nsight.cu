
/*
Nsight Systems：用于高层次的系统级性能分析，帮助你识别整个应用的瓶颈，例如 GPU 内核启动延迟、数据传输等问题。 
Nsight Compute：用于深入分析单个 CUDA 内核的性能瓶颈，帮助你优化内核代码。
通常，你可以使用 Nsight Systems 先找到瓶颈的 CUDA 内核，然后使用 Nsight Compute 对这些内核进行详细的性能分析。
一般分两步走：先在运行环境中生成报告，然后在可视化软件中查看分析报告。
Tips：早期cuda可能包含nvvp或者nvprof，这两个工具已经被废弃了，性能分析以nsight为准。
*/
#include <stdio.h>
#include <hip/hip_runtime.h>

// 向量加法 cuda kernel
__global__ void vector_add(float *a, float *b, float *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        c[i] = a[i] + b[i];
    }
}

__global__ void transposeNative(float* input, float* output, int m, int n)
{
    int colID_input = threadIdx.x + blockDim.x * blockIdx.x;
    int rowID_input = threadIdx.y + blockDim.y * blockIdx.y;
    if (rowID_input < m && colID_input < n)
    {
        int index_input = colID_input + rowID_input * n;
        int index_output = rowID_input + colID_input * m;
        output[index_output] = input[index_input];
    }
}

// 向量加法 host函数
void vector_add_host(float *a, float *b, float *c, int n)
{
    // 申请显存
    float *dev_a, *dev_b, *dev_c;
    hipMalloc((void **)&dev_a, n * sizeof(float));
    hipMalloc((void **)&dev_b, n * sizeof(float));
    hipMalloc((void **)&dev_c, n * sizeof(float));

    // 将数据从主机内存拷贝到显存
    hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    // 启动kernel
    transposeNative<<<(n+127)/128, 128>>>(dev_a, dev_b, 10000, 10000);
    vector_add<<<(n + 255) / 256, 256>>>(dev_a, dev_b, dev_c, n);
    vector_add<<<(n + 255) / 256, 256>>>(dev_a, dev_b, dev_c, n);
    vector_add<<<(n + 255) / 256, 256>>>(dev_a, dev_b, dev_c, n);

    // 将数据从显存拷贝到主机内存
    hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost);

    // 释放显存
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}

// 矩阵转置 host函数
void transpose_host(float* input, float* output, int m, int n)
{
    // 申请显存
    float* dev_input, *dev_output;
    hipMalloc((void**)&dev_input, m * n * sizeof(float));
    hipMalloc((void**)&dev_output, m * n * sizeof(float));

    // 将数据从主机内存拷贝到显存
    hipMemcpy(dev_input, input, m * n * sizeof(float), hipMemcpyHostToDevice);

    // 启动kernel
    dim3 block(16, 16);
    dim3 grid((n + block.x - 1) / block.x, (m + block.y - 1) / block.y);
    transposeNative<<<grid, block>>>(dev_input, dev_output, m, n);

    // 将数据从显存拷贝到主机内存
    hipMemcpy(output, dev_output, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // 释放显存
    hipFree(dev_input);
    hipFree(dev_output);
}

int main()
{
    int n = 100000000;
    float *a = new float[n];
    float *b = new float[n];
    float *c = new float[n];

    for (int i = 0; i < n; i++)
    {
        a[i] = 100.0f;
        b[i] = 200.0f;
    }

    int loop = 1000;
    while (loop--){
        vector_add_host(a, b, c, n);
    }

    delete[] a;
    delete[] b;
    delete[] c;
    printf("Done\n");
    return 0;
}


