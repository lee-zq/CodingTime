#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <string>
#include <stdio.h>
// #include <hip/hip_runtime.h>
// #include <>

__global__ void default_kernel()
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    printf("device print: {[gridDim=(%d,%d,%d), blockDim=(%d,%d,%d)] [blockIdx=(%d,%d,%d), threadIdx=(%d,%d,%d)] "
           "[global_tid=%d]}\n",
           gridDim.x,
           gridDim.y,
           gridDim.z,
           blockDim.x,
           blockDim.y,
           blockDim.z,
           blockIdx.x,
           blockIdx.y,
           blockIdx.z,
           threadIdx.x,
           threadIdx.y,
           threadIdx.z,
           tid);
}

int test_cuda_kernel()
{
    /*
    kernel编写和调用方式
    1. 单独编写kernel，使用<<<grid, block>>>启动内核；
    2. 使用默认流启动kernel时，使用<<<grid, block>>>启动内核；
    3. 使用非空流启动kernel时，使用<<<grid, block, 0, stream>>>启动内核；
    4. kernel函数对于主机线程是异步的，需要显式调用hipDeviceSynchronize()等待kernel执行完成；
    */

    // 创建一个网格和一个块，类型是dim3，在默认流中启动内核
    dim3 grid(10, 1, 1);
    dim3 block(10, 1, 1);
    // Launch the kernel
    default_kernel<<<grid, block>>>();
    hipDeviceSynchronize();
    printf("test1 Done\n");
    return 0;
}

int test_cuda_stream()
{
    /*
    cuda流的创建和使用：
    1. 创建非空流，一般使用hipStreamCreate(&stream)；
    2. 创建带flag的非空流，一般使用hipStreamCreateWithFlags(&stream, flag)，flag可以用于指定该流和默认流之间的同步策略；
    3. 不同流之间的操作是不保证时序的，同一流内的操作是保证时序的；
    */

    dim3 grid(10, 1, 1);
    dim3 block(10, 1, 1);
    // 1. 直接创建非空流
    hipStream_t stream1;
    hipStreamCreate(&stream1);

    // 2. 带flag的非空流
    // 2.1 使用hipStreamDefault，我理解是和默认流隐式，与默认流交替同步执行，与上述hipStreamCreate一样
    hipStream_t stream21;
    hipStreamCreateWithFlags(&stream21, hipStreamDefault);
    // 2.2 使用hipStreamNonBlocking，表示流是异步的，与默认流并行执行
    hipStream_t stream22;
    hipStreamCreateWithFlags(&stream22, hipStreamNonBlocking);

    default_kernel<<<grid, block, 0, stream1>>>();
    default_kernel<<<grid, block, 0, stream21>>>();
    default_kernel<<<grid, block, 0, stream22>>>();
    hipDeviceSynchronize();

    printf("test2 Done\n");
    return 0;
}


int test_cuda_copy()
{
    /*
    cuda拷贝函数的使用：
    1. 默认同步拷贝函数实际是在默认流上执行的，对于主机线程是阻塞式的；
    2. 非空流异步拷贝函数是在指定流上执行的，对于主机线程是非阻塞式的；
    3. 流内操作是保证时序的, 可以将多个操作提交到一个流后，再等待执行完成，多个流之间是不保证时序的；
    */

    // 创建host内存并初始化
    int mem_size = 1024 * 10;
    int* host_mem = (int*)malloc(mem_size * sizeof(int));
    for (int i = 0; i < mem_size; i++)
    {
        host_mem[i] = i;
    }
    // 1. 默认流同步执行。接口是同步的，但是实际上是异步的，因为hipMemcpy是阻塞的，会等待拷贝完成
    int* device_mem1;
    hipMalloc(&device_mem1, mem_size * sizeof(int));
    hipMemcpy(device_mem1, host_mem, mem_size * sizeof(int), hipMemcpyHostToDevice);
    // do something on device
    hipMemcpy(host_mem, device_mem1, mem_size * sizeof(int), hipMemcpyDeviceToHost);

    // 2. 非空流异步执行。接口是异步的，实际上是异步的，因为hipMemcpyAsync是非阻塞的，host线程不会等待拷贝完成,需要显式调用hipDeviceSynchronize()等待拷贝完成
    int* device_mem2;
    hipMalloc(&device_mem2, mem_size * sizeof(int));
    hipStream_t stream2;
    hipStreamCreate(&stream2);
    hipMemcpyAsync(device_mem2, host_mem, mem_size * sizeof(int), hipMemcpyHostToDevice, stream2);
    hipDeviceSynchronize();
    // do something on device
    hipMemcpyAsync(host_mem, device_mem2, mem_size * sizeof(int), hipMemcpyDeviceToHost, stream2);
    hipDeviceSynchronize();

    // 释放内存、流对象、设备内存
    free(host_mem);
    hipFree(device_mem1);
    hipFree(device_mem2);
    hipStreamDestroy(stream2);

    printf("test3 Done\n");
    return 0;
}

int test_cuda_event()
{
    /*
    cuda事件的使用：
    1. hipEventCreate(&event)创建事件；
    2. hipEventRecord(event, stream)记录事件，stream为空则在默认流上记录；
    3. hipEventSynchronize(event)等待事件完成；
    4. hipEventElapsedTime(&time, start, end)计算两个事件之间的时间差；
    5. hipEventDestroy(event)销毁事件；
    */

    // 创建host内存并初始化
    int mem_size = 1024 * 10;
    int* host_mem = (int*)malloc(mem_size * sizeof(int));
    for (int i = 0; i < mem_size; i++)
    {
        host_mem[i] = i;
    }
    // 创建device内存
    int* device_mem;
    hipMalloc(&device_mem, mem_size * sizeof(int));

    // 创建事件
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    // 记录事件
    hipEventRecord(start);
    hipMemcpy(device_mem, host_mem, mem_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(host_mem, device_mem, mem_size * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(end);

    // 等待事件完成
    hipEventSynchronize(end);

    // 计算时间差
    float time;
    hipEventElapsedTime(&time, start, end);
    printf("Time: %f ms\n", time);

    // 释放内存、事件对象、设备内存
    free(host_mem);
    hipFree(device_mem);
    hipEventDestroy(start);
    hipEventDestroy(end);

    printf("test4 Done\n");
    return 0;
}

int test_cuda_stream_sync()
{
    /*
    cuda流的同步：
    1. hipStreamSynchronize(stream)等待流上的所有操作完成, 相比于hipDeviceSynchronize()更细粒度，可以只等待一个流上的操作完成；
    2. hipStreamWaitEvent(stream, event)等待事件完成后再执行流上的操作；
    3. hipStreamQuery(stream)查询流上的操作是否完成；
    */

    // 创建host内存并初始化
    int mem_size = 1024 * 100;
    int* host_mem = (int*)malloc(mem_size * sizeof(int));
    for (int i = 0; i < mem_size; i++)
    {
        host_mem[i] = i;
    }
    // 创建device内存
    int* device_mem;
    hipMalloc(&device_mem, mem_size * sizeof(int));

    // 创建流和事件
    hipStream_t stream;
    hipStreamCreate(&stream);
    hipEvent_t event;
    hipEventCreate(&event);

    // 记录事件
    hipMemcpyAsync(device_mem, host_mem, mem_size * sizeof(int), hipMemcpyHostToDevice, stream);
    hipEventRecord(event, stream);
    hipMemcpyAsync(host_mem, device_mem, mem_size * sizeof(int), hipMemcpyDeviceToHost, stream);

    // 等待流上的操作完成
    hipStreamSynchronize(stream);

    // 释放内存、事件对象、设备内存
    free(host_mem);
    hipFree(device_mem);
    hipStreamDestroy(stream);
    hipEventDestroy(event);

    printf("test5 Done\n");
    return 0;
}

int test_cuda_error()
{
    /*
    cuda错误处理接口
    CUDA常见的错误码有：
    1. hipSuccess：操作成功；
    2. hipErrorIllegalAddress：无效的地址, 访存越界 code=700；
    3. hipErrorOutOfMemory：内存分配失败；
    4. hipErrorNotInitialized：初始化错误；
    5. hipErrorLaunchFailure：内核启动失败；

    */

    // hipGetErrorName函数接受一个错误码，返回错误名称
    __host__ __device__ const char* hipGetErrorName(hipError_t error);
    // hipGetErrorString函数接受一个错误码，返回错误描述
    __host__ __device__ const char* hipGetErrorString(hipError_t error);
    // hipGetLastError函数返回上一次CUDA runtime调用时的错误码，并将CUDA错误码置为hipSuccess
    __host__ __device__ hipError_t hipGetLastError(void);
    // hipPeekAtLastError函数与上面功能一样，但不会重置CUDA的错误信息
    __host__ __device__ hipError_t hipPeekAtLastError(void);

    return 0;
}


int main()
{
    int loop = 1000;
    while (loop--){
        test_cuda_kernel();
        test_cuda_stream();
        test_cuda_copy();
        test_cuda_event();
        test_cuda_stream_sync();
    }

    return 0;
}