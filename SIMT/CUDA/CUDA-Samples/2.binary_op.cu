#include "hip/hip_runtime.h"
/*
cuda实现矩阵加法
*/
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime.h>

#include "../../../base_func/timer.hpp"

__global__ void vec_add_kernel_v1(float* dev_a_ptr, float* dev_b_ptr, float* dev_c_ptr, int n)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("tid: %d, val=%.2f\n", tid, dev_c_ptr[tid]);
    if (tid < n)
    {
        dev_c_ptr[tid] = dev_a_ptr[tid] * dev_b_ptr[tid];
    }
}

void vec_add_func(float* host_a_ptr, float* host_b_ptr, float* host_c_ptr, int element_size)
{
    float* dev_a_ptr;
    float* dev_b_ptr;
    float* dev_c_ptr;
    int mem_size = element_size * sizeof(float);
    hipMalloc((void**)&dev_a_ptr, mem_size);
    hipMalloc((void**)&dev_b_ptr, mem_size);
    hipMalloc((void**)&dev_c_ptr, mem_size);

    int loop = 1000;
    while (loop--)
    {
        // Timer t;
        hipMemcpy(dev_a_ptr, host_a_ptr, mem_size, hipMemcpyKind::hipMemcpyHostToDevice);
        hipMemcpy(dev_b_ptr, host_b_ptr, mem_size, hipMemcpyKind::hipMemcpyHostToDevice);
        vec_add_kernel_v1<<< element_size / 512 + 1 , 512>>>(dev_a_ptr, dev_b_ptr, dev_c_ptr, element_size);
        hipDeviceSynchronize();
        hipMemcpy(host_c_ptr, dev_c_ptr, mem_size, hipMemcpyKind::hipMemcpyDeviceToHost);
    }

    hipFree(dev_a_ptr);
    hipFree(dev_b_ptr);
    hipFree(dev_c_ptr);
}

int main()
{
    int element_size = 1024 * 1024;
    float* host_a_ptr = (float*)malloc(sizeof(float) * element_size);
    float* host_b_ptr = (float*)malloc(sizeof(float) * element_size);
    float* host_c_ptr = (float*)malloc(sizeof(float) * element_size);
    for (size_t i = 0; i < element_size; i++)
    {
        host_a_ptr[i] = 100.f;
        host_b_ptr[i] = 200.f;
    }
    
    vec_add_func(host_a_ptr, host_b_ptr, host_c_ptr, element_size);

    // for (size_t i = 0; i < element_size; i++)
    // {
    //     if (host_c_ptr[i] - 300.f > 0.01f){
    //         printf("Error at idx=%zd , val=%.2f\n", i, host_c_ptr[i]);
    //         std::abort();
    //     }
    // }

    delete host_a_ptr;
    delete host_b_ptr;
    delete host_c_ptr;

    printf("Success.\n");
    return 0;
}