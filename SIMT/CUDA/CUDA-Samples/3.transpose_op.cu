#include "hip/hip_runtime.h"
/*
cuda实现element wise op
*/
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime.h>

#include "../../../base_func/timer.hpp"

// 后向索引版本，线程配置以输入张量shape排布，计算每个线程对应的数据源位置和目标位置
__global__ void Transpose_kernel_v1(float* dev_a_ptr, float* dev_b_ptr, int m, int n)
{
    const int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
    const int colIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // printf("rowIdx: %d, colIdx=%d\n", rowIdx, colIdx);
    if (rowIdx < m && colIdx < n)
    {
        dev_b_ptr[colIdx * m + rowIdx] = dev_a_ptr[rowIdx * n + colIdx];
    }
}

// 前向索引版本，线程配置以输出张量shape排布，计算每个线程对应的数据目标位置和原位置
// cuda编程多以这种形式申请计算资源，以一个目标值为单位划分一个线程进行计算，特点是目标值之间不关联
__global__ void Transpose_kernel_v2(float* dev_a_ptr, float* dev_b_ptr, int m, int n)
{
    const int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
    const int colIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // printf("rowIdx: %d, colIdx=%d\n", rowIdx, colIdx);
    if (rowIdx < n && colIdx < m)
    {
        dev_b_ptr[rowIdx * m + colIdx] = dev_a_ptr[colIdx * n + rowIdx];
    }
}

__global__ void Transpose_kernel_v3(float* dev_a_ptr, float* dev_b_ptr, int m, int n)
{
    const int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
    const int colIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // printf("rowIdx: %d, colIdx=%d\n", rowIdx, colIdx);
    if (rowIdx < n && colIdx < m)
    {
        dev_b_ptr[rowIdx * m + colIdx] = dev_a_ptr[colIdx * n + rowIdx];
    }
}

void mat_transpose_op_func(float* host_a_ptr, float* host_b_ptr, int m, int n)
{
    float* dev_a_ptr;
    float* dev_b_ptr;
    int mem_size = m * n * sizeof(float);
    hipMalloc((void**)&dev_a_ptr, mem_size);
    hipMalloc((void**)&dev_b_ptr, mem_size);

    std::vector<int> block_len_x = {4, 8, 16, 32};
    std::vector<int> block_len_y = {4, 8, 16, 32};
    for (auto x : block_len_x)
    {
        for (auto y : block_len_y)
        {

            dim3 block(x, y);
            // dim3 grid(n / x + 1, m / y + 1); // v1配置 src->dst
            dim3 grid((m + x - 1) / x, (n + y - 1) / y); // v2配置 dst->src
            int loop = 10;
            while (loop--)
            {
                // Timer t;
                hipMemcpy(dev_a_ptr, host_a_ptr, mem_size, hipMemcpyKind::hipMemcpyHostToDevice);
                // Transpose_kernel_v1<<<grid, block>>>(dev_a_ptr, dev_b_ptr, m, n);
                Transpose_kernel_v2<<<grid, block>>>(dev_a_ptr, dev_b_ptr, m, n);

                hipDeviceSynchronize();
                hipMemcpy(host_b_ptr, dev_b_ptr, mem_size, hipMemcpyKind::hipMemcpyDeviceToHost);
            }
        }
    }

    hipFree(dev_a_ptr);
    hipFree(dev_b_ptr);
}

int main()
{
    int m = 2300;
    int n = 1500;
    int element_size = m * n;
    float* host_a_ptr = (float*)malloc(sizeof(float) * element_size);
    float* host_b_ptr = (float*)malloc(sizeof(float) * element_size);
    // int idx = 0;
    // for (size_t i = 0; i < m; i++)
    // {
    //     for (size_t j = 0; j < n; j++)
    //     {
    //         host_a_ptr[i * n + j] = ++idx;
    //         printf("%.2f, ", host_a_ptr[i * n + j]);
    //     }
    //     std::cout << std::endl;
    // }
    std::cout << std::endl;

    mat_transpose_op_func(host_a_ptr, host_b_ptr, m, n);

    // for (size_t i = 0; i < n; i++)
    // {
    //     for (size_t j = 0; j < m; j++)
    //     {
    //         printf("%.2f, ", host_b_ptr[i * m + j]);
    //     }
    //     std::cout << std::endl;
    // }
    std::cout << std::endl;

    delete host_a_ptr;
    delete host_b_ptr;

    printf("Success.\n");
    return 0;
}