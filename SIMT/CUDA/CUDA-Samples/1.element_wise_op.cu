#include "hip/hip_runtime.h"
/*
cuda实现element wise op
*/
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime.h>

#include "../../../base_func/timer.hpp"

__global__ void Relu_kernel_v1(float* dev_a_ptr, float* dev_b_ptr, int n)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("tid: %d, val=%.2f\n", tid, dev_c_ptr[tid]);
    if (tid < n)
    {
        if (dev_a_ptr[tid] > 0)
        {
            dev_b_ptr[tid] = dev_a_ptr[tid];
        }else{
            dev_b_ptr[tid] = 0;
        }
    }
}

void vec_element_wise_op_func(float* host_a_ptr, float* host_b_ptr, int element_size)
{
    float* dev_a_ptr;
    float* dev_b_ptr;
    int mem_size = element_size * sizeof(float);
    hipMalloc((void**)&dev_a_ptr, mem_size);
    hipMalloc((void**)&dev_b_ptr, mem_size);

    int loop = 1000;
    while (loop--)
    {
        // Timer t;
        hipMemcpy(dev_a_ptr, host_a_ptr, mem_size, hipMemcpyKind::hipMemcpyHostToDevice);
        Relu_kernel_v1<<<element_size / 512 + 1, 512>>>(dev_a_ptr, dev_b_ptr, element_size);
        hipDeviceSynchronize();
        hipMemcpy(host_b_ptr, dev_b_ptr, mem_size, hipMemcpyKind::hipMemcpyDeviceToHost);
    }

    hipFree(dev_a_ptr);
    hipFree(dev_b_ptr);
}

int main()
{
    int element_size = 1024 * 1024;
    float* host_a_ptr = (float*)malloc(sizeof(float) * element_size);
    float* host_b_ptr = (float*)malloc(sizeof(float) * element_size);
    for (size_t i = 0; i < element_size; i++)
    {
        if (i%2==0){
            host_a_ptr[i] = 100.f;
        }{
            host_a_ptr[i] = -100.f;
        }
    }

    vec_element_wise_op_func(host_a_ptr, host_b_ptr, element_size);

    for (size_t i = 0; i < element_size; i++)
    {
        if (host_b_ptr[i] - std::max(0.0f, host_a_ptr[i]) < 0.0f){
            printf("Error at idx=%zd , val=%.2f\n", i, host_b_ptr[i]);
            std::abort();
        }
    }

    delete host_a_ptr;
    delete host_b_ptr;

    printf("Success.\n");
    return 0;
}